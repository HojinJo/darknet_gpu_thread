#include "hip/hip_runtime.h"
#include "hiprand/hiprand.h"
#include "hipblas.h"

extern "C" {

}

__global__ void network_num(int index){
    network_num<<<index,index>>>(index);
}