#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "hip/hip_runtime.h"
}

__global__ void network_num_kernel(int index){
    int i =0;
    int sum =0;
    for(i=0;i<index*100;i++){
        sum *= index;
    }
}

void network_num(int index){
    /*if(index == 0)
    network_num_kernel<<<1,1>>>(index);
    else if(index == 1)
    network_num_kernel<<<2,2>>>(index);
    else if(index == 2)
    network_num_kernel<<<3,3>>>(index);
    else if(index == 3)
    network_num_kernel<<<4,4>>>(index);
    else if(index == 2)
    network_num_kernel<<<2,2>>>(index);
    else if(index == 2)
    network_num_kernel<<<2,2>>>(index);*/
    network_num_kernel<<<cuda_grid_test(index),1>>>(index);
    check_error(hipPeekAtLastError());
}