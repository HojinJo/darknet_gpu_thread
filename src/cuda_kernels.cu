#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "hip/hip_runtime.h"
}

__global__ void network_num_kernel(int index){
    int sum = 1;
    int sum += 1;
}

void network_num(int index){
    network_num_kernel<<<index,index>>>(index);
    check_error(hipPeekAtLastError());
}