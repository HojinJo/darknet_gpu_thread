#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "hip/hip_runtime.h"
}

__global__ void network_num_kernel(int index){
    int i =0,j=0;
    int sum =0;
    for(i=0;i<index*1000;i++){
        sum += 1;
        for(j=0;j<1000;j++)
            sum *= index;
    }
}

void network_num(int index,hipStream_t *stream){
    network_num_kernel<<<cuda_grid_test(index),1,0,*stream>>>(index);
    check_error(hipPeekAtLastError());
}